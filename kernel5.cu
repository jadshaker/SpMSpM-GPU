#include "hip/hip_runtime.h"
#include "common.h"

#define BLOCK_DIM 64

__global__ void spmspm_kernel5(COOMatrix *cooMatrix1,
                               CSRMatrix *csrMatrix1,
                               CSCMatrix *cscMatrix1,
                               COOMatrix *cooMatrix2,
                               CSRMatrix *csrMatrix2,
                               CSCMatrix *cscMatrix2,
                               COOMatrix *cooMatrix3,
                               const unsigned int numRows1,
                               const unsigned int numRows2,
                               const unsigned int numCols2,
                               const unsigned int numNonzeros1,
                               const unsigned int numNonzeros2)
{
    extern __shared__ float row[];
    __shared__ unsigned int nnz;

    if (threadIdx.x == 0)
    {
        nnz = 0;
    }

    for (int i = threadIdx.x; i < numCols2; i += blockDim.x)
    {
        row[i] = 0;
    }
    __syncthreads();

    unsigned int rowA = blockIdx.x;
    unsigned int rowStart1 = csrMatrix1->rowPtrs[rowA];
    unsigned int rowEnd1 = csrMatrix1->rowPtrs[rowA + 1];

    for (unsigned int i = rowStart1 + threadIdx.x; i < rowEnd1; i += blockDim.x)
    {
        float valA = csrMatrix1->values[i];
        unsigned int rowB = csrMatrix1->colIdxs[i];

        unsigned int rowStart2 = csrMatrix2->rowPtrs[rowB];
        unsigned int rowEnd2 = csrMatrix2->rowPtrs[rowB + 1];

        for (unsigned int j = rowStart2; j < rowEnd2; ++j)
        {
            unsigned int colB = csrMatrix2->colIdxs[j];
            float valB = csrMatrix2->values[j];
            float val = valA * valB;
            if (val != 0.0f)
            {
                float oldVal = atomicAdd(&row[colB], val);
                if (oldVal == 0.0f)
                {
                    atomicAdd(&nnz, 1);
                }
            }
        }
    }
    __syncthreads();

    if (nnz != 0)
    {
        __shared__ unsigned int idx;
        if (threadIdx.x == 0)
        {
            idx = atomicAdd(&cooMatrix3->numNonzeros, nnz);
        }
        __syncthreads();

        for (int i = threadIdx.x; i < numCols2; i += blockDim.x)
        {
            float v = row[i];
            if (v != 0.0f)
            {
                unsigned int index = atomicAdd(&idx, 1);
                cooMatrix3->rowIdxs[index] = rowA;
                cooMatrix3->colIdxs[index] = i;
                cooMatrix3->values[index] = v;
            }
        }
    }
}

void spmspm_gpu5(COOMatrix *cooMatrix1,
                 CSRMatrix *csrMatrix1,
                 CSCMatrix *cscMatrix1,
                 COOMatrix *cooMatrix2,
                 CSRMatrix *csrMatrix2,
                 CSCMatrix *cscMatrix2,
                 COOMatrix *cooMatrix3,
                 unsigned int numRows1,
                 unsigned int numRows2,
                 unsigned int numCols2,
                 unsigned int numNonzeros1,
                 unsigned int numNonzeros2)
{
    const dim3 block(BLOCK_DIM);
    const dim3 grid(numRows1);

    spmspm_kernel5<<<grid, block, numCols2 * sizeof(float)>>>(cooMatrix1,
                                                              csrMatrix1,
                                                              cscMatrix1,
                                                              cooMatrix2,
                                                              csrMatrix2,
                                                              cscMatrix2,
                                                              cooMatrix3,
                                                              numRows1,
                                                              numRows2,
                                                              numCols2,
                                                              numNonzeros1,
                                                              numNonzeros2);
}